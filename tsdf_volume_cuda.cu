#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 *  Author: Marsette Vona
 */

#include "device_cuda.h"

using namespace rxkinfu::device;

namespace rxkinfu
{
  namespace device
  {
    template<typename T>
    __global__ void
    initializeVolume (PtrStep<T> volume)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      
      if (x < VOLUME_X && y < VOLUME_Y)
      {
          T *pos = volume.ptr(y) + x;
          int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
          for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step)
             pack_tsdf (0.f, 0, *pos);
      }
    }   
  }
}

void
rxkinfu::device::initVolume (PtrStep<short2> volume)
{
  dim3 block (32, 16);
  dim3 grid (1, 1, 1);
  grid.x = divUp (VOLUME_X, block.x);      
  grid.y = divUp (VOLUME_Y, block.y);

  initializeVolume<<<grid, block>>>(volume);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace rxkinfu
{
  namespace device
  {
    struct Tsdf
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
        MAX_WEIGHT = 1 << 7
      };

      mutable PtrStep<short2> volume;
      float3 cell_size;

      Intr intr;

      Mat33 Rcurr_inv;
      float3 tcurr;

      PtrStepSz<ushort> depth_raw; //depth in mm

      float trunc_dist_mm;

      __device__ __forceinline__ float3
      getVoxelGCoo (int x, int y, int z) const
      {
        float3 coo = make_float3 (x, y, z);
        coo += 0.5f;         //shift to cell center;

        coo.x *= cell_size.x;
        coo.y *= cell_size.y;
        coo.z *= cell_size.z;

        return coo;
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
          return;

        short2 *pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(*pos);

        for (int z = 0; z < VOLUME_Z; ++z, pos += elem_step)
        {
          float3 v_g = getVoxelGCoo (x, y, z);            //3 // p

          //transform to curr cam coo space
          float3 v = Rcurr_inv * (v_g - tcurr);           //4

          int2 coo;           //project to current cam
          coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
          coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);

          if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)           //6
          {
            int Dp = depth_raw.ptr (coo.y)[coo.x];

            if (Dp != 0)
            {
              float xl = (coo.x - intr.cx) / intr.fx;
              float yl = (coo.y - intr.cy) / intr.fy;
              float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

              float sdf = 1000 * norm (tcurr - v_g) * lambda_inv - Dp; //mm

              sdf *= (-1);

              if (sdf >= -trunc_dist_mm)
              {
                float tsdf = fmin (1, sdf / trunc_dist_mm);

                int weight_prev;
                float tsdf_prev;

                //read and unpack
                unpack_tsdf (*pos, tsdf_prev, weight_prev);

                const int Wrk = 1;

                float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                int weight_new = min (weight_prev + Wrk, MAX_WEIGHT);

                pack_tsdf (tsdf_new, weight_new, *pos);
              }
            }
          }
        }
      }
    };

    __global__ void
    integrateTsdfKernel (const Tsdf tsdf) {
      tsdf ();
    }

    __global__ void
    tsdf2 (PtrStep<short2> volume, const float trunc_dist_mm, const Mat33 Rcurr_inv, float3 tcurr,
           const Intr intr, const PtrStepSz<ushort> depth_raw, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      short2 *pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_x = Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z;
      float v_y = Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z;
      float v_z = Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z;

//#pragma unroll
      for (int z = 0; z < VOLUME_Z; ++z)
      {
        float3 vr;
        vr.x = v_g_x;
        vr.y = v_g_y;
        vr.z = (v_g_z + z * cell_size.z);

        float3 v;
        v.x = v_x + Rcurr_inv.data[0].z * z * cell_size.z;
        v.y = v_y + Rcurr_inv.data[1].z * z * cell_size.z;
        v.z = v_z + Rcurr_inv.data[2].z * z * cell_size.z;

        int2 coo;         //project to current cam
        coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
        coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);


        if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)         //6
        {
          int Dp = depth_raw.ptr (coo.y)[coo.x]; //mm

          if (Dp != 0)
          {
            float xl = (coo.x - intr.cx) / intr.fx;
            float yl = (coo.y - intr.cy) / intr.fy;
            float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

            float sdf = Dp - norm (vr) * lambda_inv * 1000; //mm


            if (sdf >= -trunc_dist_mm)
            {
              float tsdf = fmin (1.f, sdf / trunc_dist_mm);

              int weight_prev;
              float tsdf_prev;

              //read and unpack
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }
        pos += elem_step;
      }       /* for(int z = 0; z < VOLUME_Z; ++z) */
    }      /* __global__ */
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
rxkinfu::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size,
                                      const Mat33& Rcurr_inv, const float3& tcurr, float trunc_dist, 
                                      PtrStep<short2> volume)
{
  Tsdf tsdf;

  tsdf.volume = volume;  
  tsdf.cell_size.x = volume_size.x / VOLUME_X;
  tsdf.cell_size.y = volume_size.y / VOLUME_Y;
  tsdf.cell_size.z = volume_size.z / VOLUME_Z;
  
  tsdf.intr = intr;

  tsdf.Rcurr_inv = Rcurr_inv;
  tsdf.tcurr = tcurr;
  tsdf.depth_raw = depth_raw;

  tsdf.trunc_dist_mm = trunc_dist*1000; //mm

  dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

#if 0
   //tsdf2<<<grid, block>>>(volume, trunc_dist, Rcurr_inv, tcurr, intr, depth_raw, tsdf.cell_size);
   integrateTsdfKernel<<<grid, block>>>(tsdf);
#endif
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}


namespace rxkinfu
{
  namespace device
  {
    __global__ void
    scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= depth.cols || y >= depth.rows)
        return;

      int Dp = depth.ptr (y)[x];

      float xl = (x - intr.cx) / intr.fx;
      float yl = (y - intr.cy) / intr.fy;
      float lambda = sqrtf (xl * xl + yl * yl + 1);

      scaled.ptr (y)[x] = Dp * lambda/1000.f; //meters
    }

    __global__ void
    tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            const float trunc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float trunc_dist_inv = 1.0f / trunc_dist;

      short2* pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if (Dp_scaled != 0 && sdf >= -trunc_dist) //meters
          {
            float tsdf = fmin (1.0f, sdf * trunc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, *pos);
          }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__

    __global__ void
    tsdf23normal_hack (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                  const float trunc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

        const float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        const float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        float trunc_dist_inv = 1.0f / trunc_dist;

        short2* pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(short2);

        //#pragma unroll
        for (int z = 0; z < VOLUME_Z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled,
            pos += elem_step)
        {
            float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
            if (inv_z < 0)
                continue;

            // project to current cam
            int2 coo =
            {
                __float2int_rn (v_x * inv_z + intr.cx),
                __float2int_rn (v_y * inv_z + intr.cy)
            };

            if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
            {
                float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

                float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

                if (Dp_scaled != 0 && sdf >= -trunc_dist) //meters
                {
                    float tsdf = fmin (1.0f, sdf * trunc_dist_inv);                                              

                    bool integrate = true;
                    if ((x > 0 &&  x < VOLUME_X-2) && (y > 0 && y < VOLUME_Y-2) && (z > 0 && z < VOLUME_Z-2))
                    {
                        const float qnan = numeric_limits<float>::quiet_NaN();
                        float3 normal = make_float3(qnan, qnan, qnan);

                        float Fn, Fp;
                        int Wn = 0, Wp = 0;
                        unpack_tsdf (*(pos + elem_step), Fn, Wn);
                        unpack_tsdf (*(pos - elem_step), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.z = (Fn - Fp)/cell_size.z;

                        unpack_tsdf (*(pos + volume.step/sizeof(short2) ), Fn, Wn);
                        unpack_tsdf (*(pos - volume.step/sizeof(short2) ), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.y = (Fn - Fp)/cell_size.y;

                        unpack_tsdf (*(pos + 1), Fn, Wn);
                        unpack_tsdf (*(pos - 1), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.x = (Fn - Fp)/cell_size.x;

                        if (normal.x != qnan && normal.y != qnan && normal.z != qnan)
                        {
                            float norm2 = dot(normal, normal);
                            if (norm2 >= 1e-10)
                            {
                                normal *= rsqrt(norm2);

                                float nt = v_g_x * normal.x + v_g_y * normal.y + v_g_z * normal.z;
                                float cosine = nt * rsqrt(v_g_x * v_g_x + v_g_y * v_g_y + v_g_z * v_g_z);

                                if (cosine < 0.5)
                                    integrate = false;
                            }
                        }
                    }

                    if (integrate)
                    {
                        //read and unpack
                        float tsdf_prev;
                        int weight_prev;
                        unpack_tsdf (*pos, tsdf_prev, weight_prev);

                        const int Wrk = 1;

                        float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                        int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                        pack_tsdf (tsdf_new, weight_new, *pos);
                    }
                }
            }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
rxkinfu::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                      const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                      float trunc_dist,
                                      PtrStep<short2> volume, DeviceArray2D<float>& depthScaled)
{
  depthScaled.create (depth.rows, depth.cols);

  dim3 block_scale (32, 8);
  dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

  //scales depth along ray and converts mm -> meters. 
  scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
  cudaSafeCall ( hipGetLastError () );

  float3 cell_size;
  cell_size.x = volume_size.x / VOLUME_X;
  cell_size.y = volume_size.y / VOLUME_Y;
  cell_size.z = volume_size.z / VOLUME_Z;

  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 block (16, 16);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  tsdf23<<<grid, block>>>(depthScaled, volume, trunc_dist, Rcurr_inv, tcurr, intr, cell_size);    
  //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, trunc_dist, Rcurr_inv, tcurr, intr, cell_size);

  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace rxkinfu {
  namespace device {

    __global__ void
    shiftTsdf(const int3 trans,
              PtrStep<short2> from_volume, PtrStep<short2> to_volume) { 

      const int x = threadIdx.x + blockIdx.x * blockDim.x;
      const int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y) return;

      const int from_x = x + trans.x, from_y = y + trans.y;
      int from_z = trans.z;

      const int from_step = from_volume.step * VOLUME_Y / sizeof(short2);
      const int to_step = to_volume.step * VOLUME_Y / sizeof(short2);

      short2* from_pos = from_x + from_volume.ptr (from_y) + from_z * from_step;
      short2* to_pos = x + to_volume.ptr (y);

      const bool skip = (from_x < 0 || from_y < 0 ||
                         from_x >= VOLUME_X || from_y >= VOLUME_Y);

      short2 sentinel, *from;
      pack_tsdf (0.0f, 0, sentinel);

      for(int z = 0; z < VOLUME_Z; ++z, ++from_z){

        if( skip || from_z < 0 || from_z >= VOLUME_Z ) from = &sentinel;
        else from = from_pos;
        
        __syncthreads(); //saves about a ms

        *to_pos = *from;

        from_pos += from_step;
        to_pos += to_step;
      }
    }
  }
}

void rxkinfu::device::shiftTsdfVolume(const int3& trans,
                                      PtrStep<short2> from_volume,
                                      PtrStep<short2> to_volume) {
  dim3 block (VOLUME_X, 1);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  shiftTsdf<<<grid, block>>>(trans, from_volume, to_volume);    

  cudaSafeCall (hipGetLastError());
  cudaSafeCall (hipDeviceSynchronize());
}

namespace rxkinfu {
  namespace device {

    /** \brief Linearly interpolate tsdf values.
      *
      * \param[in] t the interpolation value in [0,1]
      * \param[in] a the first tsdf value pair
      * \param[in] b the second tsdf value pair
      * \param[out] r the interpolated distance and weight are stored here
      * \param[in] checkvalid check weights for invalids
      */
    template<bool checkvalid>
    __device__ __forceinline__ void
    linterp(float t, const float2 &a, const float2 &b, float2 &r) {

      float ad = a.x, aw = a.y, bd = b.x, bw = b.y;

      if (!checkvalid || ((aw != 0) && (bw != 0)) || ((aw == 0) && (bw == 0))) {
        r.x = (1-t) * ad + t * bd;
        r.y = (1-t) * aw + t * bw;
      } else if ((aw != 0) && (t < 0.5f)) {
        r.x = ad;
        r.y = aw;
      } else if ((bw != 0) && (t > 0.5f)) {
        r.x = bd;
        r.y = bw;
      } else {
        r.x = 0;
        r.y = 0;
      }
    }

    /** \brief Linearly interpolate tsdf cells.
      *
      * \param[in] t the interpolation value in [0,1]
      * \param[in] a the first tsdf cell
      * \param[in] b the second tsdf cell
      * \param[out] r the interpolated distance and weight are stored here
      * \param[in] checkvalid check weights for invalids
      */
    template<bool checkvalid>
    __device__ __forceinline__ void
   linterp(float t, const short2 &a, const short2 &b, float2 &r) {

      float ad, bd;
      int aw, bw;

      unpack_tsdf(a, ad, aw);
      unpack_tsdf(b, bd, bw);

      linterp<checkvalid>(t, make_float2(ad, aw), make_float2(bd, bw), r);
    }


    /** \brief trilinearly interpolate in a tsdf volume.
     *
     * \param[in] from_volume the volume in which to interpolate
     * \param[in] from_step z-step in from_volume
     * \param[in] point the 3D point at which to interpolate in grid cell units
     * \param[in] checknn first check if nearest neighbor is a sentinel
     * \param[out] tsdf the interpolated distance value is written here
     * \param[out] tsdf the interpolated weight is written here
     */
    template<bool checknn, bool checkvalid>
    __device__ __forceinline__ void
    trilinterp(PtrStep<short2> &from_volume, const int from_step,
               const float3 &point, float &tsdf, int &weight) {

      int3 g, g2;
      short2 *from_pos;
      float3 diff;
      float2 buf[4];

      //default to uninitialized
      tsdf = 0.0f; weight = 0;

      //grid coords of neighborhood upper left corner
      g.x = __float2int_rd (point.x);
      g.y = __float2int_rd (point.y);
      g.z = __float2int_rd (point.z);
          
      //it's about 10ms faster to check the whole neighborhood than to check
      //each neighbor addr
      if (g.x >= 0 && g.y >= 0 && g.z >= 0 &&
          g.x < VOLUME_X - 1 && g.y < VOLUME_Y - 1 && g.z < VOLUME_Z - 1) {
        
        if (checknn) {
          
          //first check if nn is empty
          
          //seems to save an extra 5-10ms 
          
          //while it seems like this would mean 5 total fetches if the nn
          //is not empty, it is not a win to cache the nn value explicitly
          //here, probably it will be in cache when it is
          //read again below anyway.
          
          g2.x = __float2int_rn (point.x);
          g2.y = __float2int_rn (point.y);
          g2.z = __float2int_rn (point.z);
          
          from_pos = g2.x + from_volume.ptr (g2.y) + g2.z * from_step;
          unpack_tsdf(*from_pos, tsdf, weight);
       }

        if ((!checknn) || ((weight != 0) && (tsdf != 1.0f))) {
          
          //interpolate in 2x2x2 neighborhood
          
          diff.x = point.x - g.x;
          diff.y = point.y - g.y;
          diff.z = point.z - g.z;
              
          for (int i = 0; i < 4; i++) {
            from_pos = 
              g.x + 
              from_volume.ptr (g.y + (i&1)) +
              + (g.z + ((i>>1)&1)) * from_step;
            linterp<checkvalid>(diff.x, *from_pos, *(from_pos+1), buf[i]);
          }
              
          for (int i = 0; i < 2; i++)
            linterp<checkvalid>(diff.y, buf[2*i], buf[2*i+1], buf[i]);
          
          linterp<checkvalid>(diff.z, buf[0], buf[1], buf[0]);
          
          tsdf = buf[0].x;
          weight = min(__float2int_rn (buf[0].y), Tsdf::MAX_WEIGHT);
          
          //clamp normalized tsdf
          if(tsdf >= 0.0f) tsdf = fmin(1.0f, tsdf);
          else tsdf = fmax(-1.0f, tsdf);
        }
      }
    }
  }
}

namespace rxkinfu {
  namespace device {

    template<bool checknn, bool checkvalid>
    __device__ __forceinline__ void
    transformTsdf(const float3 trans, const Mat33 rot,
                  PtrStep<short2> from_volume,
                  PtrStep<short2> to_volume) {

      // Graveyard of attempted optimizations:
      //
      // * save 2x2x2 neighborhood in a thread-local buffer, then for z>0 shift
      //   the buffer geometrically and fetch only the newly opened cells
      //
      // * various shared memory explicit caching schemes
      //
      // * nearest-neighbor only (fast but quality terrible)
      //
      // * nearest and 2nd nearest neighbor only (quality terrible)
  
      const int x = threadIdx.x + blockIdx.x * blockDim.x;
      const int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      short2 *to_pos = to_volume.ptr (y) + x;

      const int from_step = from_volume.step * VOLUME_Y / sizeof(short2);
      const int to_step = to_volume.step * VOLUME_Y / sizeof(short2);

      float3 point;

      float tsdf = 0.0f;
      int weight = 0;

      for(int z = 0; z < VOLUME_Z; ++z) {

        point = (rot * make_float3(x, y, z)) + trans;

        trilinterp<checknn, checkvalid>(from_volume, from_step, point,
                   tsdf, weight);

        pack_tsdf (tsdf, weight, *to_pos);

        to_pos += to_step;
      }
    }

    __global__ void
    transformTsdfFF(const float3 trans, const Mat33 rot,
                    PtrStep<short2> from_volume,
                    PtrStep<short2> to_volume) {
      transformTsdf<false,false>(trans, rot, from_volume, to_volume);
    }

    __global__ void
    transformTsdfFT(const float3 trans, const Mat33 rot,
                    PtrStep<short2> from_volume,
                    PtrStep<short2> to_volume) {
      transformTsdf<false,true>(trans, rot, from_volume, to_volume);
    }

    __global__ void
    transformTsdfTF(const float3 trans, const Mat33 rot,
                    PtrStep<short2> from_volume,
                    PtrStep<short2> to_volume) {
      transformTsdf<true,false>(trans, rot, from_volume, to_volume);
    }

    __global__ void
    transformTsdfTT(const float3 trans, const Mat33 rot,
                    PtrStep<short2> from_volume,
                    PtrStep<short2> to_volume) {
      transformTsdf<true,true>(trans, rot, from_volume, to_volume);
    }
  }
}

void
rxkinfu::device::transformTsdfVolume(const float3& trans, const Mat33& rot,
                                     PtrStep<short2> from_volume,
                                     PtrStep<short2> to_volume,
                                     const bool checknn,
                                     const bool checkvalid) {
  dim3 block (VOLUME_X, 1);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  if (!checknn && !checkvalid)
    transformTsdfFF<<<grid, block>>>(trans, rot, from_volume, to_volume);
  else if (!checknn && checkvalid)
    transformTsdfFT<<<grid, block>>>(trans, rot, from_volume, to_volume);
  else if (checknn && !checkvalid)
    transformTsdfTF<<<grid, block>>>(trans, rot, from_volume, to_volume);
  else if (checknn && checkvalid)
    transformTsdfTT<<<grid, block>>>(trans, rot, from_volume, to_volume);

  cudaSafeCall (hipGetLastError());
  cudaSafeCall (hipDeviceSynchronize());
}

